#include "hip/hip_runtime.h"
#define N_TRIAL 100000
#define N_SAMPLES 1000

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <ctime>

#include <thrust/unique.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/set_operations.h>

#include "Graph.hpp"
#include "RuleReader.hpp"
#include "TesttripleReader.hpp"
#include "ScoreTree.hpp"
#include "Rule.hpp"

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__device__ void applyCyclicRule(int* adj_list_starts, int* adj_lists, int * val, int* relations, int rulelength, int N, int * result, int * resultlength, int& counter, int * previous) {
	counter++;
	if (counter == N_TRIAL) {
		return;
	}
	int value = *val;
	previous[N] = value;
	int * adj_list = &(adj_lists[adj_list_starts[*relations]]);
	int start_indptr = 3;
	int size_indptr = adj_list[1];
	int start_ind = start_indptr + size_indptr;
	//int size_ind = adj_list[2];

	relations++;
	N++;
	int ind_ptr = adj_list[start_indptr + value];
	int len = adj_list[start_indptr + value + 1] - ind_ptr;
	if (N == rulelength) {
		for (int j = 0; j < len; j++)
		{
			int nextval = adj_list[start_ind + ind_ptr + j];
			for (int i = 0; i < N; i++) {
				if (previous[i] == nextval) {
					break;
				}
			}
			result[*resultlength] = nextval;
			(*resultlength)++;
			if (*resultlength == N_SAMPLES) return;
			counter++;
			if (counter == N_TRIAL) {
				return;
			}
		}
	}
	else {
		for (int j = 0; j < len; j++)
		{
			int nextval = adj_list[start_ind + ind_ptr + j];
			for (int i = 0; i < N; i++) {
				if (previous[i] == nextval) {
					break;
				}
			}
			applyCyclicRule(adj_list_starts, adj_lists, &nextval, relations, rulelength, N, result, resultlength, counter, previous);
			if (*resultlength == N_SAMPLES) return;
		}
	}
}
__device__ void applyAcyclicRule(int* adj_list_starts, int* adj_lists, int* relations, int rulelength, int N, int * result, int * resultlength, int& counter, int * previous) {
	int * adj_list = &(adj_lists[adj_list_starts[*relations]]);
	int start_indptr = 3;
	int size_indptr = adj_list[1];
	int start_ind = start_indptr + size_indptr;
	//int size_ind = adj_list[2];

	relations++;
	N++;

	if (N == rulelength) {
		for (int val = 0; val < size_indptr; val++) {
			previous[N - 1] = val;
			int ind_ptr = adj_list[start_indptr + val];
			int len = adj_list[start_indptr + val + 1] - ind_ptr;
			for (int j = 0; j < len; j++) {
				int nextval = adj_list[start_ind + ind_ptr + j];
				for (int i = 0; i < N; i++) {
					if (previous[i] == nextval) {
						break;
					}
				}
				result[*resultlength] = nextval;
				(*resultlength)++;
				if (*resultlength == N_SAMPLES) return;
				counter++;
				if (counter == N_TRIAL) {
					return;
				}
			}
		}
	}
	else {
		for (int val = 0; val < size_indptr; val++) {
			previous[N - 1] = val;
			int ind_ptr = adj_list[start_indptr + val];
			int len = adj_list[start_indptr + val + 1] - ind_ptr;
			for (int j = 0; j < len; j++) {
				int nextval = adj_list[start_ind + ind_ptr + j];
				for (int i = 0; i < N; i++) {
					if (previous[i] == nextval) {
						break;
					}
				}
				applyCyclicRule(adj_list_starts, adj_lists, &nextval, relations, rulelength, N, result, resultlength, counter, previous);
				if (*resultlength == N_SAMPLES) return;
			}
		}
	}
}
__device__ bool existsAcyclic(int* adj_list_starts, int* adj_lists, int * valId, int * constant, int* relations, int N) {
	int * adj_list = &(adj_lists[adj_list_starts[*relations]]);
	int start_indptr = 3;
	int size_indptr = adj_list[1];
	int start_ind = start_indptr + size_indptr;
	//int size_ind = adj_list[2];

	relations++;
	N--;
	int ind_ptr = adj_list[start_indptr + *valId];
	int len = adj_list[start_indptr + *valId + 1] - ind_ptr;
	if (N == 0 && constant != nullptr) {
		for (int j = 0; j < len; j++) {
			int to = adj_list[start_ind + ind_ptr + j];
			if (to == *constant) {
				return true;
			}
		}
		return false;
	}
	else if (N == 0 && constant == nullptr) {
		if (len > 0) {
			return true;
		}
		else {
			return false;
		}
	}
	else {
		for (int j = 0; j < len; j++) {
			if (existsAcyclic(adj_list_starts, adj_lists, &adj_list[start_ind + ind_ptr + j], constant, relations, N)) {
				return true;
			};
		}
		return false;
	}
}

__device__ void computeHeadsCyclic(int* adj_list_starts, int* adj_lists, int * valId, Rule& rule, int * result, int * resultlength, int * previous) {
	int* relations = rule.getRelationsBwd();
	int counter = 0;
	applyCyclicRule(adj_list_starts, adj_lists, valId, relations, rule.getRulelength(), 0, result, resultlength, counter, previous);
}
__device__ void computeTailsCyclic(int* adj_list_starts, int* adj_lists, int * valId, Rule& rule, int * result, int * resultlength, int * previous) {
	int* relations = rule.getRelationsFwd();
	int counter = 0;
	applyCyclicRule(adj_list_starts, adj_lists, valId, relations, rule.getRulelength(), 0, result, resultlength, counter, previous);
}
__device__ void computeAcyclic(int* adj_list_starts, int* adj_lists, Rule& rule, int * result, int * resultlength, int * previous) {
	int* relations = rule.getRelationsBwd();
	int counter = 0;
	if (*(rule.getBodyconstantId()) != INT_MAX) {
		applyCyclicRule(adj_list_starts, adj_lists, rule.getBodyconstantId(), relations, rule.getRulelength(), 0, result, resultlength, counter, previous);
	}
	else {
		applyAcyclicRule(adj_list_starts, adj_lists, relations, rule.getRulelength(), 0, result, resultlength, counter, previous);
	}
}
__device__ bool existsAcyclic(int* adj_list_starts, int* adj_lists, int * valId, Rule& rule) {
	int* relations = rule.getRelationsFwd();
	int * constantnode = nullptr;
	if (*(rule.getBodyconstantId()) != INT_MAX) {
		constantnode = rule.getBodyconstantId();
	}
	return existsAcyclic(adj_list_starts, adj_lists, valId, constantnode, relations, rule.getRulelength());
}

__global__
void run(int* adj_list_starts, int* adj_lists, int * rulesAdjBegin, Rule * rulesAdjList, int** testtriples, int * testtriplessize, int ** results, int * resultlengths, int ** previous, ScoreTree * sts) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	int * adj_begin = rulesAdjBegin;
	Rule * rules_adj_list = rulesAdjList;
	//Foreach testtriples
	for (int i = index; index < *testtriplessize; i += stride) {
		sts[i].initializeScoreTree();

		int * testtriple = testtriples[i];
		int ind_ptr = adj_begin[3 + testtriple[1]];
		int len = adj_begin[3 + testtriple[1] + 1] - ind_ptr;
		// Foreach rule
		for (int j = 0; j < len; j++) {
			Rule currRule = rules_adj_list[ind_ptr + j];
			//printf("Rule ");
			//currRule.print();
			//printf("\n");
			//currRule.toString();
			//std::cout << "Testtriple: " << *testtriple[0] << std::endl;
			if (currRule.getRuletype() == Ruletype::XRule) {
				if (existsAcyclic(adj_list_starts, adj_lists, &testtriple[0], currRule)) {
					results[i][resultlengths[i]] = testtriple[0];
				}
				//std::cout << "exists: " << x << std::endl;
			}
			else if (currRule.getRuletype() == Ruletype::YRule) {
				if (*currRule.getHeadconstant() == testtriple[0]) {
					computeAcyclic(adj_list_starts, adj_lists, currRule, results[i], &resultlengths[i], previous[i]);
					/*
					std::cout << "Reslen: " << *resultlength << std::endl;
					for (int a = 0; a < *resultlength; a++) {
						std::cout <<"acsol: "<< *results[a] << std::endl;
					}
					*/
				}
			}
			else {
				computeTailsCyclic(adj_list_starts, adj_lists, &testtriple[0], currRule, results[i], &resultlengths[i], previous[i]);
				/*
				std::cout << "Reslen: " << *resultlength << std::endl;
				for (int a = 0; a < *resultlength; a++) {
					std::cout << "cycsol: " << *results[a] << std::endl;
				}
				*/
			}


			//Unique values of results
			if (resultlengths[i] > 0) {
				thrust::sort(thrust::device, results[i], (results[i] + resultlengths[i]));
				int * end = thrust::unique(thrust::device, results[i], (results[i] + resultlengths[i]));
				//Filter results from trainingsset
				int * adj_list = &(adj_lists[adj_list_starts[testtriple[1]]]);
				int * indptr = &adj_list[3 + testtriple[0]];
				int len = *(indptr + 1) - *indptr;
				int * ind = &adj_list[3 + adj_list[1] + *indptr];
				thrust::sort(thrust::device, ind, ind + len);
				int * endDiff = thrust::set_difference(thrust::device, results[i], end, ind, ind + len, results[i]);
				int nValues = thrust::distance(results[i], endDiff);

				sts[i].addValues(currRule.getAppliedConfidence(), results[i], nValues);
				resultlengths[i] = 0;
			}

		}
		sts[i].Free();
	}
}

int main()
{
	size_t limit = 7000000000;
	hipDeviceSetLimit(hipLimitMallocHeapSize,limit); 
	unsigned int intermediate = clock();

	//"C:\\Users\\Simon\\Desktop\\Parullel_Cuda_wAdj\\train.txt"
	Graph * graph = new Graph("/home/eisernried12/train.txt", true);
	int * nodesAdjLists = graph->getAdjLists();
	int * nodesAdjListStarts = graph->getAdjListStarts();
	printf("Graph created in millisecs: %ld\n", clock() - intermediate);
	intermediate = clock();

	
	//"C:\\Users\\Simon\\Desktop\\Parullel_Cuda_wAdj\\rules.txt"
	RuleReader rr = RuleReader("/home/eisernried12/alpha-50", graph, true);
	int * rulesAdjBegin = rr.getRuleAdjBegin();
	Rule * rulesAdjList = rr.getRulesAdjList();
	printf("Rules created in millisecs: %ld\n", clock() - intermediate);
	intermediate = clock();

	//"C:\\Users\\Simon\\Desktop\\Parullel_Cuda_wAdj\\test.txt"
	TesttripleReader ttr = TesttripleReader("/home/eisernried12/test.txt", graph,true);
	int ** testtriples = ttr.getTesttriples();
	int * testtriplesSize = ttr.getTesttriplesSize();
	printf("Testtriples created in millisecs: %ld\n", clock() - intermediate);
	intermediate = clock();

	int ** results;
	int * resultlengths;
	int ** previous;
	ScoreTree * sts;
	hipMallocManaged(&results, *testtriplesSize * sizeof(int*));
	hipMallocManaged(&resultlengths, *testtriplesSize * sizeof(int));
	hipMallocManaged(&previous, *testtriplesSize * sizeof(int*));
	hipMallocManaged(&sts, *testtriplesSize * sizeof(ScoreTree));
	for (int i = 0; i < *testtriplesSize; i++) {
		hipMallocManaged(&results[i], N_SAMPLES * sizeof(int));
		hipMallocManaged(&previous[i], 10 * sizeof(int));
		resultlengths[i] = 0;
	}
	cudaCheckErrors("ScoreTree");

	int blockSize = 256;
	int numBlocks = (*testtriplesSize + blockSize - 1) / blockSize;

	std::cout << "START" << std::endl;
	run <<<numBlocks, blockSize>>> (nodesAdjListStarts, nodesAdjLists, rulesAdjBegin, rulesAdjList, testtriples, testtriplesSize, results, resultlengths, previous, sts);
	
	hipDeviceSynchronize();
	
	std::cout << "Rules applied in millisecs: " << clock() - intermediate << std::endl;
	hipDeviceReset();
	
	return 0;
}
